#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "kernels.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if(err != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

__global__ void convolution_kernel(const float* input, const float* kernel, float* output,
                                 int input_width, int kernel_size, int output_width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < output_width && col < output_width) {
        float sum = 0.0f;
        for (int i = 0; i < kernel_size; ++i) {
            for (int j = 0; j < kernel_size; ++j) {
                int input_row = row + i;
                int input_col = col + j;
                sum += input[input_row * input_width + input_col] * 
                      kernel[i * kernel_size + j];
            }
        }
        output[row * output_width + col] = sum;
    }
}

__global__ void relu_kernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output[idx] = fmaxf(0.0f, input[idx]);
}

__global__ void softmax_kernel(float* input, float* output, int size) {
    float max_val = input[0];
    for (int i = 1; i < size; ++i)
        max_val = fmaxf(max_val, input[i]);

    float sum = 0.0f;
    for (int i = 0; i < size; ++i)
        sum += expf(input[i] - max_val);

    for (int i = 0; i < size; ++i)
        output[i] = expf(input[i] - max_val) / sum;
}

__global__ void fc_kernel(const float* input, const float* weights, const float* bias,
                        float* output, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        float sum = 0.0f;
        for (int i = 0; i < input_size; ++i)
            sum += input[i] * weights[idx * input_size + i];
        output[idx] = sum + bias[idx];
    }
}

// Wrappers
extern "C" {
    void convolution_forward(float* d_input, float* d_kernel, float* d_output,
            int input_width, int kernel_size, int output_width) {
        if (!d_input || !d_kernel || !d_output) {
        printf("Error: Null pointer detected!\n");
        return;
        }

        printf("input_width: %d, kernel_size: %d, output_width: %d\n",
        input_width, kernel_size, output_width);

        dim3 block(16, 16);
        dim3 grid((output_width + block.x - 1) / block.x,
        (output_width + block.y - 1) / block.y);

        printf("Grid: (%d, %d), Block: (%d, %d)\n",
        grid.x, grid.y, block.x, block.y);

        convolution_kernel<<<grid, block>>>(d_input, d_kernel, d_output,
                            input_width, kernel_size, output_width);
        CUDA_CHECK(hipGetLastError());
    }

    void relu_activation(float* d_input, float* d_output, int size) {
        dim3 block(256);
        dim3 grid((size + block.x - 1) / block.x);
        relu_kernel<<<grid, block>>>(d_input, d_output, size);
        CUDA_CHECK(hipGetLastError());
    }

    void softmax(float* d_input, float* d_output, int size) {
        softmax_kernel<<<1, 1>>>(d_input, d_output, size);
        CUDA_CHECK(hipGetLastError());
    }

    void fc_forward(float* d_input, float* d_weights, float* d_bias,
                  float* d_output, int input_size, int output_size) {
        dim3 block(256);
        dim3 grid((output_size + block.x - 1) / block.x);
        fc_kernel<<<grid, block>>>(d_input, d_weights, d_bias, d_output,
                                 input_size, output_size);
        CUDA_CHECK(hipGetLastError());
    }
}

extern "C" {
    void* cuda_malloc(size_t size) {
        void* ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        return ptr;
    }
    
    void cuda_free(void* ptr) {
        CUDA_CHECK(hipFree(ptr));
    }
    
    void copy_to_device(void* dest, void* src, size_t size) {
        CUDA_CHECK(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
    }
    
    void copy_to_host(void* dest, void* src, size_t size) {
        if (!dest || !src) {
            printf("Error: Null pointer in copy_to_host (dest: %p, src: %p)\n", dest, src);
            return;
        }
        
        // Verify src is device pointer
        hipPointerAttribute_t attributes;
        hipError_t err = hipPointerGetAttributes(&attributes, src);
        if (err != hipSuccess) {
            printf("Error: Source pointer is not a valid CUDA pointer\n");
            return;
        }
        

            CUDA_CHECK(hipDeviceSynchronize()); // Ensure previous operations completed
            CUDA_CHECK(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
    }
}
